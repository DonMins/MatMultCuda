#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <stdio.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define BLOCK_SIZE 32
#define N 2048

#define CUDA_CHECK_ERROR(err)           \
if (err != hipSuccess) {          \
printf("Cuda error: %s\n", hipGetErrorString(err));    \
printf("Error in file: %s, line: %i\n", __FILE__, __LINE__);  \
}                 \

/**
* Метод уножения матриц на GPU
* @params: matrix A, matrix B, result matrix C
*/

__global__ void matMult(float* A, float* B, float* C){
	// Индекс блока
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Индекс нити
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	float sum = 0.0;
	//Индекс A[i][0]
	int ia = N * BLOCK_SIZE * by + N * ty;
	// Индекс B[0][j]
	int ib = BLOCK_SIZE * bx + tx;


	for (int k = 0; k < N; k++) {
		sum += A[ia + k] * B[ib + k * N];
	}
	// Индекс C[i][j]
	int ic = N * BLOCK_SIZE * by + BLOCK_SIZE * bx;

	//Результирующая матрица
	C[ic + N * ty + tx] = sum;
}


/**
* Метод уножения матриц на CPU
* @params: matrix A, matrix B, result matrix C
*/

void cpu_ikj(float* A, float* B, float* C) {
	for (int i = 0; i < N; i++) {
		for (int k = 0; k < N; k++) {
			for (int j = 0; j < N; j++) {
				C[i * N + j] += A[i * N + k] * B[k * N + j];
			}
		}
	}
}

/**
* Метод вывода матриц
* @params: matrix C
*/
void printMatrix(float* C) {
	for ( int i = 0; i < N; i++){
		for ( int j = 0; j < N; j++){
			printf("%.3f ", C[i * N + j]);
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}




int main() {
	setlocale(LC_ALL, "RUS");

	//Выделяем память под матрицы на хосте

	float *A = (float*) malloc(N * N *sizeof(float));
	float *B = (float*) malloc(N * N* sizeof(float));
	float *C_GPU = (float*) malloc(N * N *sizeof(float));
	float *C_CPU = (float*) malloc(N * N*  sizeof(float));


	// Заполняем матрицы 
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++) {
			A[i + j * N] = i + j;
			B[i + j * N] = i + j;
		}
	//printMatrix(A);
	//printMatrix(B);
	//Конфигурация запуска ядра

	dim3 dimGrid(N / BLOCK_SIZE, N / BLOCK_SIZE);//Размер используемого грида
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);//Размер используемого блока

	hipEvent_t start;
	hipEvent_t stop;

	//Создаем event'ы для синхронизации и замера времени работы GPU

	CUDA_CHECK_ERROR(hipEventCreate(&start));
	CUDA_CHECK_ERROR(hipEventCreate(&stop));


	float* adev, *bdev, *cdev;

	//Выделяем глобальную память для храния данных на девайсе
	CUDA_CHECK_ERROR(hipMalloc((void**)&adev, N * N * sizeof(float *)));
	CUDA_CHECK_ERROR(hipMalloc((void**)&bdev, N * N * sizeof(float *)));
	CUDA_CHECK_ERROR(hipMalloc((void**)&cdev, N * N * sizeof(float *)));

	//Копируем исходные матрицы с хоста на девайс
	CUDA_CHECK_ERROR(hipMemcpy(adev, A, N * N * sizeof(float *), hipMemcpyHostToDevice));
	CUDA_CHECK_ERROR(hipMemcpy(bdev, B, N * N * sizeof(float *), hipMemcpyHostToDevice));

	//Отмечаем старт расчетов на GPU
	hipEventRecord(start, 0);    
								

	matMult << < dimGrid, dimBlock >> > (adev, bdev , cdev);

	//Отмечаем окончание расчета
	hipEventRecord(stop, 0);    

	float timeGPU = 0;

	//Синхронизируемя с моментом окончания расчетов
	hipEventSynchronize(stop);   

	//Рассчитываем время работы GPU
	hipEventElapsedTime(&timeGPU, start, stop);    

	std::cout << "Время умножения матриц размером " << N << "x" << N << " на GPU = " << timeGPU << " мсек" << std::endl;

	//Копируем результат с девайса на хост
	CUDA_CHECK_ERROR(hipMemcpy(C_GPU, cdev, N * N * sizeof(float *), hipMemcpyDeviceToHost));

	//printMatrix(C_GPU);
	//Чистим ресурсы на видеокарте

	CUDA_CHECK_ERROR(hipEventDestroy(start));
	CUDA_CHECK_ERROR(hipEventDestroy(stop));

	CUDA_CHECK_ERROR(hipFree(adev));
	CUDA_CHECK_ERROR(hipFree(bdev));
	CUDA_CHECK_ERROR(hipFree(cdev));

	double start_time = clock();

	cpu_ikj(A, B, C_CPU);

	double end_time = clock();

	std::cout << "Время умножения матриц размером " << N << "x" << N << " на CPU = " << ((end_time - start_time)) *1000 / CLOCKS_PER_SEC << " мсек" << std::endl;
	
	//Чистим память на хосте
	delete A;
	delete B;
	delete C_GPU;
	delete C_CPU;
	system("pause");
	return 0;
}
